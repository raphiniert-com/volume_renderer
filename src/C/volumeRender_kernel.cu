#include "hip/hip_runtime.h"
/*! \file volumeRender_kernel.cu
 * 	\author Raphael Scheible <raphael.scheible@uniklinik-freiburg.de>
 * 	\version 1.0
 * 	\license This project is released under the GNU Affero General Public License, Version 3
 *
 * 	\brief CUDA C file with all the device functions
 *
 */

#ifndef _VOLUMERENDER_KERNEL_CU_
#define _VOLUMERENDER_KERNEL_CU_

#include <float.h>
#include <hip/hip_vector_types.h>
#include <math.h>
#include <stdio.h>
#include <vector>
#include <volumeRender.h>

#define ONE_OVER_2PI ((float)0.1591549430918953357688837633725143620344596457404564)
#define PI2 ((float)6.2831853071795864769252867665590057683943387987502116)

/*! \var typedef unsigned int uint
 * 	\brief defines abbrev for unsigned int: uint
 */
typedef unsigned int uint;

/*! \var typedef unsigned char uchar
 * 	\brief  defines abbrev for unsigned char: uchar
 */
typedef unsigned char uchar;

/*! \var typedef float3 (*gradientFunction)(const float3, const float3,
 * 				const float3, const float3, const float3, const float3) 
 *  \brief function pointer to gradientFunction that returns a gradient
 */
typedef float3 (*gradientFunction)(const float3, const float3, const float3,
                                   const float3, const float3, const float3);
/*! \enum gradientMethod
 * 	\brief possible gradient computation methods
 */
enum gradientMethod {
  gradientCompute = 0, /*!< gradient computation on the fly */
  gradientLookup = 1   /*!< use LUT to estimate gradient */
};

// forward declaration
__device__ float3 computeGradient(const float3, const float3, const float3,
                                  const float3, const float3, const float3);

__device__ float3 lookupGradient(const float3, const float3, const float3,
                                 const float3, const float3, const float3);

/*! \var __device__ gradientFunction gradient_functions[2] = { computeGradient, lookupGradient }; 
 *  \brief Contains function pointer of possible lookup functions
 */
__device__ gradientFunction gradient_functions[2] = {computeGradient,
                                                     lookupGradient};

/*! \var __device__ __constant__ gradientMethod dc_activeGradientMethod
 * 	\brief current chosen gradient Method. Default value is gradientCompute.
 */
__device__ __constant__ gradientMethod dc_activeGradientMethod =
    gradientCompute;

/*! \var vr::LightSource *d_lightSources
 * 	\brief device array of lightsources
 */
vr::LightSource *d_lightSources = NULL;

/*! \var __device__ __constant__ size_t c_numLightSources
 * 	\brief device variable storing number of lightsources
 */
__device__ __constant__ size_t c_numLightSources;

/*! \var hipArray *d_emissionArray
 * 	\brief device array of the emission volume data
 */
hipArray *d_emissionArray = 0;

/*! \var hipArray *d_gradientXArray
 * 	\brief device array of the gradient volume data in x direction
 */
hipArray *d_gradientXArray = 0;

/*! \var hipArray *d_gradientYArray
 * 	\brief device array of the gradient volume data in y direction
 */
hipArray *d_gradientYArray = 0;

/*! \var hipArray *d_gradientZArray
 * 	\brief device array of the gradient volume data in z direction
 */
hipArray *d_gradientZArray = 0;

/*! \var hipArray *d_absorptionArray
 * 	\brief device array of the absorption volume data
 */
hipArray *d_absorptionArray = 0;

/*! \var hipArray *d_reflectionArray
 * 	\brief device array of the reflection volume data
 */
hipArray *d_reflectionArray = 0;

/*! \var hipArray *d_illuminationArray
 * 	\brief device array of the illumination volume data
 */
hipArray *d_illuminationArray = 0;

/*! \var texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> tex_emission 
 * \brief 3D texture for emission lookup
 */
texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> tex_emission;

/*! \var texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> tex_gradientX 
 *  \brief 3D texture of gradient in x direction used in lookupGradient
 */
texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> tex_gradientX;

/*! \var texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> tex_gradientY 
 * \brief 3D texture of gradient in y direction used in lookupGradient
 */
texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> tex_gradientY;

/*! \var texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> tex_gradientZ 
 *  \brief 3D texture of gradient in z direction used in lookupGradient
 */
texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> tex_gradientZ;

/*! \var texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> tex_absorption 
 * \brief 3D texture for absorption lookup
 */
texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> tex_absorption;

/*! \var texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> tex_reflection 
 *  \brief 3D texture for reflection lookup
 */
texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> tex_reflection;

/*! \var texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> tex_illumination 
 *  \brief 3D texture for illumination lookup
 */
texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> tex_illumination;

/*! \fn int intersectBox(Ray aRay, float3 aBoxmin, float3 aBoxmax, float *aTnear, float *aTfar) 
 * \brief Intersect ray with a box. (see https://doi.org/10.1080/2151237X.2005.10129188) 
 * \param aRay ray tested for intersection. 
 * \param aBoxmin min box coordinates. 
 * \param aBoxmax max box coordinates. 
 * \param aTnear tnear plane. 
 * \param aTfar tfar plane. 
 * \return 1 if the intersects the box, 0 if not
 */
__forceinline__ __device__ int intersectBox(vr::Ray aRay, float3 aBoxmin,
                                            float3 aBoxmax, float *aTnear,
                                            float *aTfar) {
  int sign[3];
  float3 parameters[2] = {aBoxmin, aBoxmax};
  float3 inv_direction = 1.f / aRay.direction;
  float3 origin = aRay.origin;

  sign[0] = (inv_direction.x < 0);
  sign[1] = (inv_direction.y < 0);
  sign[2] = (inv_direction.z < 0);

  // intersection computation
  float tmin, tmax, tymin, tymax, tzmin, tzmax;

  tmin = (parameters[sign[0]].x - origin.x) * inv_direction.x;
  tmax = (parameters[1 - sign[0]].x - origin.x) * inv_direction.x;
  tymin = (parameters[sign[1]].y - origin.y) * inv_direction.y;
  tymax = (parameters[1 - sign[1]].y - origin.y) * inv_direction.y;

  if ((tmin > tymax) || (tymin > tmax))
    return false;

  if (tymin > tmin)
    tmin = tymin;

  if (tymax < tmax)
    tmax = tymax;

  tzmin = (parameters[sign[2]].z - origin.z) * inv_direction.z;
  tzmax = (parameters[1 - sign[2]].z - origin.z) * inv_direction.z;
  if ((tmin > tzmax) || (tzmin > tmax))
    return false;

  if (tzmin > tmin)
    tmin = tzmin;

  if (tzmax < tmax)
    tmax = tzmax;

  *aTnear = tmin;
  *aTfar = tmax;

  return true;
}

/*! \fn float3 computeGradient(const float3, const float3, const float3,
                                const float3, const float3, const float3)
 *  \brief computes gradient using central differences
 *  \param aSamplePosition position for the texture lookup [unused]
 *  \param aPosition ray position in world coordinate
 * 	\param aStep step size to a neighbor voxel
 * 	\param aBoxmin min extents of the intersection box
 * 	\param aBoxmin max extents of the intersection box
 * 	\param aBoxScale 1 devided by size of the box
 * 	\return gradient
 */
__device__ float3 computeGradient(const float3 aSamplePosition,
                                  const float3 aPosition, const float3 aStep,
                                  const float3 aBoxmin, const float3 aBoxmax,
                                  const float3 aBoxScale) {
  float3 gradient;
  float3 samplePosition1, samplePosition2;

  // normal computation using central differences
  samplePosition1 =
      (aPosition + make_float3(aStep.x, 0, 0) - aBoxmin) * (aBoxScale);
  samplePosition2 =
      (aPosition - make_float3(aStep.x, 0, 0) - aBoxmin) * (aBoxScale);

  gradient.x = tex3D(tex_emission, samplePosition1.x, samplePosition1.y,
                     samplePosition1.z) -
               tex3D(tex_emission, samplePosition2.x, samplePosition2.y,
                     samplePosition2.z);

  samplePosition1 =
      (aPosition + make_float3(0, aStep.y, 0) - aBoxmin) * (aBoxScale);
  samplePosition2 =
      (aPosition - make_float3(0, aStep.y, 0) - aBoxmin) * (aBoxScale);
  gradient.y = tex3D(tex_emission, samplePosition1.x, samplePosition1.y,
                     samplePosition1.z) -
               tex3D(tex_emission, samplePosition2.x, samplePosition2.y,
                     samplePosition2.z);

  samplePosition1 =
      (aPosition + make_float3(0, 0, aStep.z) - aBoxmin) * (aBoxScale);
  samplePosition2 =
      (aPosition - make_float3(0, 0, aStep.z) - aBoxmin) * (aBoxScale);
  gradient.z = tex3D(tex_emission, samplePosition1.x, samplePosition1.y,
                     samplePosition1.z) -
               tex3D(tex_emission, samplePosition2.x, samplePosition2.y,
                     samplePosition2.z);

  gradient = gradient * make_float3(0.5f);

  return gradient;
}

/*! \fn float3 lookupGradient (const float3, const float3, const float3,
                               const float3, const float3, const float3)
 *  \brief determines the gradient via texture lookup in gradient textures
 *  \param aSamplePosition position for the texture lookup
 *  \param aPosition ray position in world coordinate [unused]
 * 	\param aStep step size to a neighbor voxel [unused]
 * 	\param aBoxmin min extents of the intersection box [unused]
 * 	\param aBoxmin max extents of the intersection box [unused]
 * 	\param aBoxScale 1 devided by size of the box [unused]
 * 	\return gradient
 */
__device__ float3 lookupGradient(const float3 aSamplePosition,
                                 const float3 aPosition, const float3 aStep,
                                 const float3 aBoxmin, const float3 aBoxmax,
                                 const float3 aBoxScale) {
  return make_float3(tex3D(tex_gradientX, aSamplePosition.x, aSamplePosition.y,
                           aSamplePosition.z),
                     tex3D(tex_gradientY, aSamplePosition.x, aSamplePosition.y,
                           aSamplePosition.z),
                     tex3D(tex_gradientZ, aSamplePosition.x, aSamplePosition.y,
                           aSamplePosition.z));
}

/*! \fn float angle(const float3& a, const float3& b)
 *  \brief computes the angle of two vectors
 *  \param a vector a
 *  \param b vector b
 * 	\return angle between a and b
 */
__forceinline__ __device__ float angle(const float3 &a, const float3 &b) {
  // radian to degree
  return acos(dot(a, b) / (length(a) + length(b)));
}

/*! \fn float3 shade(const float3& aSamplePosition, const float3 aPosition,
                     const float3 aGradientStep, const float3 aViewPosition,
                     const float3 aColor, vr::LightSource * aLightSources, 
                     const float aScaleReflection, const float3 aBoxmin, 
                     const float3 aBoxmax, const float3 aBoxScale)
 *  \brief determines the light performed at a voxelposition of all defined lightsources
 * 			   depending on the undelying illumination texture/model
 *  \param aSamplePosition
 *  \param aPosition
 * 	\param aGradientStep step size to a neighbor voxel
 * 	\param aViewPosition position of the viewer
 * 	\param aColor the color the rendered volume absorbs
 * 	\param aLightSources pointer to all light sources
 * 	\param aScaleReflection scales the sampled value of reflection
 * 	\param aBoxmin min extents of the intersection box
 * 	\param aBoxmin max extents of the intersection box
 * 	\param aBoxScale 1 devided by size of the box
 */
__device__ float3 shade(const float3 &aSamplePosition, const float3 aPosition,
                        const float3 aGradientStep, const float3 aViewPosition,
                        const float3 aColor, vr::LightSource *aLightSources,
                        const float aScaleReflection, const float3 aBoxmin,
                        const float3 aBoxmax, const float3 aBoxScale) {
  const float scaleReflection = aScaleReflection;

  // negativ gradient approx surface normal
  const float3 surfaceNormal =
      -1 * normalize((gradient_functions[dc_activeGradientMethod])(
               aSamplePosition, aPosition, aGradientStep, aBoxmin, aBoxmax,
               aBoxScale));

  float3 result = make_float3(0.f);

  for (size_t i = 0; i < c_numLightSources; ++i) {
    vr::LightSource lightSource = aLightSources[i];

    // calculation of angles
    float3 lightPosition = (lightSource.position);
    float alpha = angle(surfaceNormal, lightPosition) / PI2 *
                  ONE_OVER_2PI; // normalizing to [0,1]
    float beta = angle(surfaceNormal, aViewPosition) / PI2 * ONE_OVER_2PI;

    float3 lightOut = (lightPosition - aPosition);
    float3 lightIn = (aViewPosition - aPosition);

    // dot( , ) here: scalar projection of lightOut/lightIn onto surfaceNormal
    float3 lightOutProj =
        lightPosition - (dot(lightOut, surfaceNormal) * surfaceNormal);
    float3 lightInProj =
        aViewPosition - (dot(lightIn, surfaceNormal) * surfaceNormal);
    float gamma = angle(lightInProj, lightOutProj) * ONE_OVER_2PI;

    // lookup in d_reflectionArray/tex_reflection
    float reflection =
        scaleReflection * tex3D(tex_reflection, aSamplePosition.x,
                                aSamplePosition.y, aSamplePosition.z);

    float light = tex3D(tex_illumination, alpha, beta, gamma);

    // consider light absorption (color of material)
    result += reflection * light * lightSource.color * aColor;
  }

  return result;
}

/*! \fn void d_render(float *d_aOutput,  const vr::RenderOptions aOptions,
         const float3 aColor, const vr::LightSource * aLightSources,
         const float3 aGradientStep)
 *  \brief performs raycasting on the device
 *  \param d_aOutput device pointer of the computed 2D output
 *  \param aOptions options of the rendering process
 * 	\param aColor the color the rendered volume absorbs
 * 	\param aLightSources pointer to all light sources
 * 	\param aGradientStep step size to a neighbor voxel
 */
__global__ void d_render(float *d_aOutput, const vr::RenderOptions aOptions,
                         const float3 aColor,
                         const vr::LightSource *aLightSources,
                         const float3 aGradientStep) {
  const uint x = blockIdx.x * blockDim.x + threadIdx.x;
  const uint y = blockIdx.y * blockDim.y + threadIdx.y;

  // return if out of image bound
  if ((x >= aOptions.image_width) || (y >= aOptions.image_height))
    return;

  vr::LightSource *lightSources =
      const_cast<vr::LightSource *>(aLightSources);
  const float tstep = aOptions.tstep;
  const float opacityThreshold = aOptions.opacity_threshold;
  const float scaleAbsorption = aOptions.scale_absorption;
  const float scaleEmission = aOptions.scale_emission;
  const float scaleReflection = aOptions.scale_reflection;

  const float3 boxMin = aOptions.boxmin;
  const float3 boxMax = aOptions.boxmax;

  // 2D image plane in [-1,1]
  float u = (x / (float)aOptions.image_width) * 2.0f - 1.0f;
  float ratio = aOptions.image_height / (float)aOptions.image_width;
  float v = (y / (float)aOptions.image_height) * 2.0f * ratio - 1.0f * ratio;

  // calculate eye ray in world space
  vr::Ray eyeRay;

  // box scale
  const float3 boxScale = 1.f / (boxMax - boxMin);

  // translate into scale
  const float cameraXOffset = aOptions.rotation_matrix.m[3].x;
  const float focalLength = aOptions.rotation_matrix.m[3].y;
  const float objectDistance = aOptions.rotation_matrix.m[3].z;

  // in case of 3D rendering we have an x offset [Off-axis]
  const float3 xVector = aOptions.rotation_matrix.m[0];
  const float3 yVector = aOptions.rotation_matrix.m[1];
  const float3 zVector = aOptions.rotation_matrix.m[2];
  const float3 vCameraOffset = (cameraXOffset * xVector);

  // Ray properties
  eyeRay.origin = vCameraOffset + (-1 * objectDistance * zVector);

  eyeRay.direction =
      normalize(u * normalize(xVector) + v * yVector + focalLength * zVector);

  // find intersection with box
  float tnear(0), tfar(0);
  int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

  // if (hit)
  //    printf("near: %f, far: %f\n", tnear, tfar);

  if (!hit)
    return;
  if (tnear < 0.0f)
    tnear = 0.0f; // clamp to near plane

  // march along ray from front to back, accumulating color
  float4 sum = make_float4(0.0f);
  float t = tnear;
  float3 pos = eyeRay.origin + eyeRay.direction * tnear;
  float3 step = eyeRay.direction * tstep;

  // map step to [0, 1] coordinates
  float3 pos_sample_old = make_float3(0.f);
  while (true) {
    // map position to [0, 1] coordinates
    float3 pos_sample = (pos - boxMin) * boxScale;

    // ################
    // ### sampling ###
    // ################

    // read from 3D texture and apply several scale factor
    float emission = scaleEmission * tex3D(tex_emission, pos_sample.x,
                                           pos_sample.y, pos_sample.z);
    float absorption = scaleAbsorption * tex3D(tex_absorption, pos_sample.x,
                                               pos_sample.y, pos_sample.z);

    float3 sample = make_float3(emission);

    // ###############################
    // ### illumination & Coloring ###
    // ###############################

    float dx = tstep;
    float alpha = 1 - __expf(-absorption * dx);

    // apply color
    float ds = tstep;
    float3 colored = sample * ds * aColor;

    float3 illumination =
        shade(pos_sample, pos, aGradientStep, eyeRay.origin, aColor,
              lightSources, scaleReflection, boxMin, boxMax, boxScale);

    float3 illuminated = colored + illumination;

    float4 shaded =
        make_float4(illuminated.x, illuminated.y, illuminated.z, alpha);

    // ###################
    // ### compositing ###
    // ###################

    // alpha-blending
    // pre-multiply alpha
    shaded.x *= shaded.w;
    shaded.y *= shaded.w;
    shaded.z *= shaded.w;

    // "under" operator for front-to-back blending
    sum = (1 - sum.w) * (shaded) + sum;

    // exit early if opaque
    if (sum.w > opacityThreshold)
      break;

    t += tstep;
    if (t > tfar)
      break;

    pos += step;
  }

  // write in image structure
  uint size = aOptions.image_width * aOptions.image_height;

// linear matlab conform memory layout (column-major)
// descibed on:
// https://eli.thegreenplace.net/2015/memory-layout-of-multi-dimensional-arrays/
#ifdef MATLAB_MEX_FILE
  uint k = x * aOptions.image_height + y;
#else
  // row-major
  uint k = y * aOptions.image_width + x;
#endif

  // write output in RBG
  d_aOutput[k] = sum.x;
  d_aOutput[k + size] = sum.y;
  d_aOutput[k + size * 2] = sum.z;
}


namespace vr {

/*! \fn hipArray* createTextureFromVolume(texture<VolumeDataType,
 *      hipTextureType3D, hipReadModeElementType>& aTex, const Volume& aVolume,
 * 			hipArray* d_aArray, const bool aNormalized=true)
 *  \brief copies Volume from host to device
 *  \param aTex texture that is used to perform lookups
 *  \param aVolume volume that is copied to the device
 * 	\param d_aArray device array where the data are stored in/copied to
 * 	\return device pointer of the device array
 */
hipArray *createTextureFromVolume(
    texture<VolumeDataType, hipTextureType3D, hipReadModeElementType> &aTex,
    const Volume &aVolume, hipArray *d_aArray) {
  // create 3D d_array
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeDataType>();
  if (d_aArray == 0)
    HANDLE_ERROR(hipMalloc3DArray(&d_aArray, &channelDesc, aVolume.extent));

  // copy data to 3D d_array
  hipMemcpy3DParms copyParams = {0};
  copyParams.srcPtr = make_hipPitchedPtr(
      aVolume.data, aVolume.extent.width * sizeof(VolumeDataType),
      aVolume.extent.width, aVolume.extent.height);
  copyParams.dstArray = d_aArray;
  copyParams.extent = aVolume.extent;
  copyParams.kind = hipMemcpyHostToDevice;
  HANDLE_ERROR(hipMemcpy3D(&copyParams));

  // set texture parameters
  aTex.normalized = true; // access with normalized texture coordinates
  aTex.filterMode = hipFilterModeLinear;     // linear interpolation
  aTex.addressMode[0] = hipAddressModeClamp; // clamp texture coordinates
  aTex.addressMode[1] = hipAddressModeClamp;
  aTex.addressMode[2] = hipAddressModeClamp;

  // bind d_aArray to 3D texture
  HANDLE_ERROR(hipBindTextureToArray(aTex, d_aArray, channelDesc));

  return d_aArray;
}

/*! \fn hipArray* setEmissionTexture(const Volume& aVolume)
 *  \brief copies emission volume from host to device
 *  \param aVolume emission volume
 */
inline hipArray *setEmissionTexture(const Volume &aVolume) {
  return createTextureFromVolume(tex_emission, aVolume, d_emissionArray);
}

/*! \fn hipArray* setAbsorptionTexture(const Volume& aVolume)
 *  \brief copies absorption volume from host to device
 *  \param aVolume absorption volume
 */
inline hipArray *setAbsorptionTexture(const Volume &aVolume) {
  return createTextureFromVolume(tex_absorption, aVolume, d_absorptionArray);
}

/*! \fn hipArray* setReflectionTexture(const Volume& aVolume)
 *  \brief copies reflection volume from host to device
 *  \param aVolume reflection volume
 */
inline hipArray *setReflectionTexture(const Volume &aVolume) {
  return createTextureFromVolume(tex_reflection, aVolume, d_reflectionArray);
}

/*! \fn hipArray* setIlluminationTexture(const Volume& aVolume)
 *  \brief copies illumination volume from host to device
 *  \param aVolume illumination volume
 */
void setIlluminationTexture(const Volume &aVolume) {
  createTextureFromVolume(tex_illumination, aVolume, d_illuminationArray);
}

/*! \fn hipArray* setGradientTextures(const Volume& aDx, const Volume& aDy, const Volume& aDz) 
 *  \brief copies gradient volumes from host to device 
 *  \param aDx volume of gradient in x direction 
 *  \param aDy volume of gradient in y direction 
 *  \param aDz volume of gradient in z direction
 */
void setGradientTextures(const Volume &aDx, const Volume &aDy,
                         const Volume &aDz) {
  createTextureFromVolume(tex_gradientX, aDx, d_gradientXArray);
  createTextureFromVolume(tex_gradientY, aDy, d_gradientYArray);
  createTextureFromVolume(tex_gradientZ, aDz, d_gradientZArray);

  // assign gradient_function
  gradientMethod tmp = gradientLookup;
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dc_activeGradientMethod), &tmp,
                                  sizeof(enum gradientMethod)));
}

/*! \fn void initCuda(const Volume& aVolumeEmission,
                      const Volume& aVolumeAbsorption,
                      const Volume& aVolumeReflection)
 *  \brief Copies volume data to device and binds textures to the appropriate data.
 *         Data of one volume can be assigned to multiple textures.
 *  \param aVolumeEmission emission volume
 *  \param aVolumeAbsorption absorption volume
 * 	\param aVolumeReflection reflection volume
 */
void initCuda(const Volume &aVolumeEmission, const Volume &aVolumeAbsorption,
              const Volume &aVolumeReflection) {
  hipArray *d_tmpEmissionArray = setEmissionTexture(aVolumeEmission);

  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeDataType>();

  if (aVolumeEmission == aVolumeAbsorption) {
#ifdef DEBUG
    printf("Emission = Absorption\n");
#endif

    HANDLE_ERROR(hipBindTextureToArray(tex_absorption, d_tmpEmissionArray,
                                        channelDesc));
  } else {
    hipArray *d_tmpAbsorptionArray = setAbsorptionTexture(aVolumeAbsorption);
    if (aVolumeAbsorption == aVolumeReflection) {
#ifdef DEBUG
      printf("Absorption = Reflection\n");
#endif
      HANDLE_ERROR(hipBindTextureToArray(tex_reflection, d_tmpAbsorptionArray,
                                          channelDesc));
    } else {
      if (aVolumeEmission == aVolumeReflection) {
#ifdef DEBUG
        printf("Emission = Reflection\n");
#endif
        HANDLE_ERROR(hipBindTextureToArray(tex_reflection, d_tmpEmissionArray,
                                            channelDesc));
      } else {
#ifdef DEBUG
        printf("All Volumes are unique\n");
#endif
        setReflectionTexture(aVolumeReflection);
      }
    }

    // no further check is necessary
    return;
  }

  // check if reflection == absorption ( == emission)
  if (aVolumeReflection == aVolumeAbsorption) {
#ifdef DEBUG
    printf("Absorption = Reflection\n");
#endif
    HANDLE_ERROR(hipBindTextureToArray(tex_absorption, d_tmpEmissionArray,
                                        channelDesc));
  } else {
    setReflectionTexture(aVolumeReflection);
  }

  return;
}

/*! \fn void freeCudaBuffers()
 *  \brief frees all device memory
 */
void freeCudaBuffers() {
  HANDLE_ERROR(hipFreeArray(d_emissionArray));
  HANDLE_ERROR(hipFreeArray(d_absorptionArray));
  HANDLE_ERROR(hipFreeArray(d_reflectionArray));

  if (d_lightSources != NULL) {
    HANDLE_ERROR(hipFree(d_lightSources));
    HANDLE_ERROR(hipFreeArray(d_illuminationArray));
  }

  // get value of dc_activeGradientMethod from device to host
  gradientMethod h_activeGradientMethod;
  hipMemcpyFromSymbol(&h_activeGradientMethod, HIP_SYMBOL(dc_activeGradientMethod),
                       sizeof(gradientMethod), 0);

  if (h_activeGradientMethod == gradientLookup) {
    HANDLE_ERROR(hipFreeArray(d_gradientXArray));
    HANDLE_ERROR(hipFreeArray(d_gradientYArray));
    HANDLE_ERROR(hipFreeArray(d_gradientZArray));
  }
}

/*! \fn void render_kernel(float* d_aOutput, const dim3& block_size,
                           const dim3& grid_size, const RenderOptions& aOptions,
                           const float3& aColor, const float3& aGradientStep)
 *  \brief starts the ray casting on the device
 * 	\param d_aOutput device pointer of the computed 2D output
 *  \param block_size CUDA block size
 * 	\param grid_size CUDA grid size
 *  \param aOptions options of the rendering process
 * 	\param aColor the color the rendered volume absorbs
 * 	\param aGradientStep step size to a neighbor voxel
 */
void render_kernel(float *d_aOutput, const dim3 &block_size,
                   const dim3 &grid_size, const RenderOptions &aOptions,
                   const float3 &aColor, const float3 &aGradientStep) {
  d_render<<<grid_size, block_size>>>(d_aOutput, aOptions, aColor,
                                      d_lightSources, aGradientStep);
}

/*! \fn void copyLightSources()
 *  \brief copy light sources to device
 * 	\param aLightSources pointer to all light sources
 * 	\param aNumOfLightSources number of light sources
 */
void copyLightSources(const LightSource *aLightSources,
                      const size_t aNumOfLightSources) {
  size_t size(aNumOfLightSources * sizeof(LightSource));
  HANDLE_ERROR(hipMalloc((void **)&d_lightSources, size));
  HANDLE_ERROR(
      hipMemcpy(d_lightSources, aLightSources, size, hipMemcpyHostToDevice));

  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(c_numLightSources), &aNumOfLightSources,
                                  sizeof(size_t)));
}

} // namespace vr
#endif // #ifndef _VOLUMERENDER_KERNEL_CU_
