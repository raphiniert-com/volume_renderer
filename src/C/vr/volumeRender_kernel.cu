#include "hip/hip_runtime.h"
/*! \file volumeRender_kernel.cu
 * 	\author Raphael Scheible <mail@raphiniert.com>
 * 	\version 1.0
 * 	\license This project is released under the GNU Affero General Public License, Version 3
 *
 * 	\brief CUDA C file with all the device functions
 */

#ifndef _VOLUMERENDER_KERNEL_CU_
#define _VOLUMERENDER_KERNEL_CU_

#include <float.h>
#include <hip/hip_vector_types.h>
#include <math.h>
#include <stdio.h>
#include <vector>
#include <vr/volumeRender.h>


#define ONE_OVER_2PI ((float)0.1591549430918953357688837633725143620344596457404564)
#define PI2 ((float)6.2831853071795864769252867665590057683943387987502116)
#define ONE_OVER_PI ((float)0.3183098861837906715377675267450287240689192914809129)
#define PI ((float)3.1415926535897932384626433832795028841971693993751058)


/*! \var typedef unsigned int uint
 * 	\brief defines abbrev for unsigned int: uint
 */
typedef unsigned int uint;

/*! \var typedef unsigned char uchar
 * 	\brief  defines abbrev for unsigned char: uchar
 */
typedef unsigned char uchar;

/*! \var typedef float3 (*gradientFunction)(const float3&, const float3&,
 * 				const float3&, const float3&, const float3&, const float3&) 
 *  \brief function pointer to gradientFunction that returns a gradient
 */
typedef float3 (*gradientFunction)(const float3&, const float3&, const float3&,
                                   const float3&, const float3&, const float3&);

/*! \var typedef float (*phaseFunction)(const float3&, const float3&, float) 
 *  \brief function pointer to a scatter function that calculates or looks up
 *         the scattering intensity based on light direction, view direction, 
 *         and an asymmetry factor.
 */
typedef float (*phaseFunction)(const float3&, const float3&, const float3&, float);                                   

// forward declaration
__device__ float3 computeGradient(const float3&, const float3&, const float3&,
                                  const float3&, const float3&, const float3&);

__device__ float3 lookupGradient(const float3&, const float3&, const float3&,
                                 const float3&, const float3&, const float3&);

__device__ float computeHG(const float3&, const float3&, const float3&, float);

__device__ float lookupPhase(const float3&, const float3&, const float3&, float);

/*! \var __device__ gradientFunction gradient_functions[2] = { computeGradient, lookupGradient }; 
 *  \brief Contains function pointer of possible gradient retrieval functions
 */
__device__ gradientFunction gradient_functions[2] = { computeGradient, lookupGradient };

/*! \var __device__ gradientFunction phase_functions[2] = { computeHG, lookupPhase }; 
 *  \brief Contains function pointer of possible scattering retrieval functions
 */
__device__ phaseFunction phase_functions[2] = { computeHG, lookupPhase };

/*! \var __device__ __constant__ GradientMethod dc_activeGradientMethod
 * 	\brief current chosen gradient Method. Default value is gradientCompute.
 */
__device__ __constant__ vr::GradientMethod dc_activeGradientMethod = vr::gradientCompute;

/*! \var __device__ __constant__ GradientMethod dc_activePhaseMethod
 * 	\brief current chosen phase Method. Default value is phaseCompute.
 */
__device__ __constant__ vr::PhaseMethod dc_activePhaseMethod = vr::phaseCompute;

/*! \var vr::LightSource *d_lightSources
 * 	\brief device array of lightsources
 */
vr::LightSource *d_lightSources = NULL;

/*! \var __device__ __constant__ size_t c_numLightSources
 * 	\brief device variable storing number of lightsources
 */
__device__ __constant__ size_t c_numLightSources;

/*! \var const hipChannelFormatDesc channelDesc
 * 	\brief channel desc for textures
 */
const hipChannelFormatDesc channelDesc = hipCreateChannelDesc<vr::VolumeDataType>();


/*! \var texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> tex_phase 
 *  \brief 3D texture for phase function lookup
 */
texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> tex_phase;

/*! \var __device__ vr::VolumeType d_idxEmmission
 * 	\brief id for the emission texture
 */
__device__ vr::VolumeType d_idxEmmission = vr::VolumeType::emission;

/*! \var __device__ vr::VolumeType d_idxAbsorption
 * 	\brief id for the absorption texture
 */
__device__ vr::VolumeType d_idxAbsorption = vr::VolumeType::emission;

/*! \var __device__ vr::VolumeType d_idxReflection
 * 	\brief id for the reflection texture
 */
__device__ vr::VolumeType d_idxReflection = vr::VolumeType::reflection;

/*! \var texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> tex_emission 
 * \brief 3D texture for emission lookup
 */
texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> tex_emission;

/*! \var texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> tex_gradientX 
 *  \brief 3D texture of gradient in x direction used in lookupGradient
 */
texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> tex_gradientX;

/*! \var texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> tex_gradientY 
 * \brief 3D texture of gradient in y direction used in lookupGradient
 */
texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> tex_gradientY;

/*! \var texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> tex_gradientZ 
 *  \brief 3D texture of gradient in z direction used in lookupGradient
 */
texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> tex_gradientZ;

/*! \var texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> tex_absorption 
 * \brief 3D texture for absorption lookup
 */
texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> tex_absorption;

/*! \var texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> tex_reflection 
 *  \brief 3D texture for reflection lookup
 */
texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> tex_reflection;

/*! \var __device__ texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> getTexture(vr::VolumeType aType)
 *  \brief function for 3D texture lookup
 *  \param aType volume type id from type vr::VolumeType
 *  \return texture given an id
 */
__device__ texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType> getTexture(vr::VolumeType aType) {
  switch (aType) {
    case vr::VolumeType::emission:
      return tex_emission;
    case vr::VolumeType::absorption:
      return tex_absorption;
    case vr::VolumeType::reflection:
      return tex_reflection;
    case vr::VolumeType::dx:
      return tex_gradientX;
    case vr::VolumeType::dy:
      return tex_gradientY;
    case vr::VolumeType::dz:
      return tex_gradientZ;
    default:
      return tex_emission;
  }
}

/*! \fn int intersectBox(Ray aRay, float3 aBoxmin, float3 aBoxmax, float *aTnear, float *aTfar) 
 * \brief Intersect ray with a box. (see https://doi.org/10.1080/2151237X.2005.10129188) 
 * \param aRay ray tested for intersection. 
 * \param aBoxmin min box coordinates. 
 * \param aBoxmax max box coordinates. 
 * \param aTnear tnear plane. 
 * \param aTfar tfar plane. 
 * \return 1 if the intersects the box, 0 if not
 */
__forceinline__ __device__ int intersectBox(vr::Ray aRay, float3 aBoxmin,
                                            float3 aBoxmax, float *aTnear,
                                            float *aTfar) {
  int sign[3];
  float3 parameters[2] = {aBoxmin, aBoxmax};
  float3 inv_direction = 1.f / aRay.direction;
  float3 origin = aRay.origin;

  sign[0] = (inv_direction.x < 0);
  sign[1] = (inv_direction.y < 0);
  sign[2] = (inv_direction.z < 0);

  // intersection computation
  float tmin, tmax, tymin, tymax, tzmin, tzmax;

  tmin = (parameters[sign[0]].x - origin.x) * inv_direction.x;
  tmax = (parameters[1 - sign[0]].x - origin.x) * inv_direction.x;
  tymin = (parameters[sign[1]].y - origin.y) * inv_direction.y;
  tymax = (parameters[1 - sign[1]].y - origin.y) * inv_direction.y;

  if ((tmin > tymax) || (tymin > tmax))
    return false;

  if (tymin > tmin)
    tmin = tymin;

  if (tymax < tmax)
    tmax = tymax;

  tzmin = (parameters[sign[2]].z - origin.z) * inv_direction.z;
  tzmax = (parameters[1 - sign[2]].z - origin.z) * inv_direction.z;
  if ((tmin > tzmax) || (tzmin > tmax))
    return false;

  if (tzmin > tmin)
    tmin = tzmin;

  if (tzmax < tmax)
    tmax = tzmax;

  *aTnear = tmin;
  *aTfar = tmax;

  return true;
}

/*! \fn float3 computeGradient(const float3, const float3, const float3,
                                const float3, const float3, const float3)
 *  \brief computes gradient using central differences
 *  \param aSamplePosition position for the texture lookup [unused]
 *  \param aPosition ray position in world coordinate
 * 	\param aStep step size to a neighbor voxel
 * 	\param aBoxmin min extents of the intersection box
 * 	\param aBoxmin max extents of the intersection box
 * 	\param aBoxScale 1 devided by size of the box
 * 	\return gradient
 */
__device__ float3 computeGradient(const float3& aSamplePosition,
                                  const float3& aPosition, const float3& aStep,
                                  const float3& aBoxmin, const float3& aBoxmax,
                                  const float3& aBoxScale) {
  float3 gradient;
  float3 samplePosition1, samplePosition2;

  // normal computation using central differences
  samplePosition1 =
      (aPosition + make_float3(aStep.x, 0, 0) - aBoxmin) * (aBoxScale);
  samplePosition2 =
      (aPosition - make_float3(aStep.x, 0, 0) - aBoxmin) * (aBoxScale);

  gradient.x = tex3D(tex_emission, samplePosition1.x, samplePosition1.y,
                     samplePosition1.z) -
               tex3D(tex_emission, samplePosition2.x, samplePosition2.y,
                     samplePosition2.z);

  samplePosition1 =
      (aPosition + make_float3(0, aStep.y, 0) - aBoxmin) * (aBoxScale);
  samplePosition2 =
      (aPosition - make_float3(0, aStep.y, 0) - aBoxmin) * (aBoxScale);

  gradient.y = tex3D(tex_emission, samplePosition1.x, samplePosition1.y,
                     samplePosition1.z) -
               tex3D(tex_emission, samplePosition2.x, samplePosition2.y,
                     samplePosition2.z);

  samplePosition1 =
      (aPosition + make_float3(0, 0, aStep.z) - aBoxmin) * (aBoxScale);
  samplePosition2 =
      (aPosition - make_float3(0, 0, aStep.z) - aBoxmin) * (aBoxScale);

  gradient.z = tex3D(tex_emission, samplePosition1.x, samplePosition1.y,
                     samplePosition1.z) -
               tex3D(tex_emission, samplePosition2.x, samplePosition2.y,
                     samplePosition2.z);

  gradient = gradient * make_float3(0.5f);

  return gradient;
}

/*! \fn float3 lookupGradient (const float3, const float3, const float3,
                               const float3, const float3, const float3)
 *  \brief determines the gradient via texture lookup in gradient textures
 *  \param aSamplePosition position for the texture lookup
 *  \param aPosition ray position in world coordinate [unused]
 * 	\param aStep step size to a neighbor voxel [unused]
 * 	\param aBoxmin min extents of the intersection box [unused]
 * 	\param aBoxmin max extents of the intersection box [unused]
 * 	\param aBoxScale 1 devided by size of the box [unused]
 * 	\return gradient
 */
__device__ float3 lookupGradient(const float3& aSamplePosition,
                                 const float3& aPosition, const float3& aStep,
                                 const float3& aBoxmin, const float3& aBoxmax,
                                 const float3& aBoxScale) {
  return make_float3(tex3D(tex_gradientX, aSamplePosition.x, aSamplePosition.y,
                           aSamplePosition.z),
                     tex3D(tex_gradientY, aSamplePosition.x, aSamplePosition.y,
                           aSamplePosition.z),
                     tex3D(tex_gradientZ, aSamplePosition.x, aSamplePosition.y,
                           aSamplePosition.z));
}

/**
 * \brief Computes the Henyey-Greenstein phase function directly.
 * 
 * \param lightDir Normalized direction of the light relative to the voxel.
 * \param viewDir  Normalized direction from the voxel to the viewer or camera.
 * \param surfaceNormal Normalized surface normal at the voxel position (unused in computation).
 * \param g        Asymmetry factor for the HG phase function.
 * \return Computed HG phase function value.
 */
__device__ float computeHG(const float3 &lightDir, const float3 &viewDir, const float3 &surfaceNormal, float g) {
    // Calculate cosTheta, the cosine of the angle between light and view directions
    float cosTheta = dot(lightDir, viewDir);
    cosTheta = fmaxf(-1.0f, fminf(1.0f, cosTheta)); // Clamp to [-1, 1]

    // Henyey-Greenstein phase function calculation
    float gSquared = g * g;
    float numerator = 1.0f - gSquared;
    float epsilon = 1e-6f; // Small value to avoid zero or negative denominator
    float denominator = powf(fmaxf(1.0f + gSquared - 2.0f * g * cosTheta, epsilon), 1.5f);

    // Handle the edge case where g is close to 1 or -1
    if (fabs(g) > 0.999f) {
        return 1.0f / (4.0f * PI);
    }

    // Final Henyey-Greenstein phase function value
    return (1.0f / (4.0f * PI)) * (numerator / denominator);
}

/**
 * \brief Looks up a precomputed HG phase function value from a 3D texture.
 * 
 * \param lightDir Normalized direction of the light relative to the voxel.
 * \param viewDir  Normalized direction from the voxel to the viewer or camera.
 * \param surfaceNormal Normalized surface normal at the voxel position.
 * \param g        Asymmetry factor parameter (unused in lookup).
 * \return Precomputed HG phase function value from the texture.
 */
__device__ float lookupPhase(const float3 &lightDir, const float3 &viewDir, const float3 &surfaceNormal, float g) {
    // Project lightDir and viewDir onto the plane orthogonal to the surfaceNormal
    float3 lightOutProj = lightDir - dot(lightDir, surfaceNormal) * surfaceNormal;
    float3 lightInProj = viewDir - dot(viewDir, surfaceNormal) * surfaceNormal;

    // Ensure the projected vectors are not zero vectors
    float lengthLightOutProj = length(lightOutProj);
    float lengthLightInProj = length(lightInProj);

    float gamma = 0.0f; // Default value for gamma
    if (lengthLightOutProj > 0.0f && lengthLightInProj > 0.0f) {
        // Normalize the projected vectors to avoid numerical instability
        lightOutProj = normalize(lightOutProj);
        lightInProj = normalize(lightInProj);

        // Calculate the angle between the two projected vectors
        gamma = acosf(fminf(fmaxf(dot(lightInProj, lightOutProj), -1.0f), 1.0f)) * ONE_OVER_2PI;
    }

    // Calculate angles alpha and beta between the vectors and the z-axis
    float alpha = acosf(fminf(fmaxf(lightDir.z, -1.0f), 1.0f)) * ONE_OVER_2PI;  // Clamp value to [-1, 1]
    float beta = acosf(fminf(fmaxf(viewDir.z, -1.0f), 1.0f)) * ONE_OVER_2PI;    // Clamp value to [-1, 1]

    // Perform texture lookup
    return tex3D<float>(tex_phase, alpha, beta, gamma);
}

/*! \fn float angle(const float3& a, const float3& b)
 *  \brief computes the angle of two vectors
 *  \param a vector a
 *  \param b vector b
 * 	\return angle between a and b
 */
__forceinline__ __device__ float angle(const float3 &a, const float3 &b) {
  // radian to degree
  float dotProd = dot(a, b) / (length(a) * length(b));
  dotProd = fminf(1.0f, fmaxf(-1.0f, dotProd)); // Clamp within [-1, 1]
  return acosf(dotProd);
}

/*! \fn float3 shade(const float3 &aSamplePosition, const float3 &aPosition, 
                     const float3 &aViewPosition, const float3 &aColor, 
                     vr::LightSource *aLightSources, const float aFactorReflection, 
                     const float3 &surfaceNormal, const float aShininess, 
                     const float aScatteringWeight, const float aHgAsymmetry)
 * \brief Calculates the illumination at a voxel position based on multiple light sources,
 *        using both Blinn-Phong reflection and Henyey-Greenstein scattering for realism.
 *
 * This function computes the shading at a specified voxel position by combining the effects
 * of Blinn-Phong reflection and Henyey-Greenstein (HG) scattering. It evaluates each light 
 * source's contribution by calculating diffuse and specular reflection from Blinn-Phong, 
 * as well as single scattering using the HG phase function. Light fall-off due to distance 
 * is incorporated using the inverse-square law. A weighting factor controls the balance 
 * between reflection and scattering.
 *
 * \param aSamplePosition 3D position of the sample within the volume, used for texture sampling.
 * \param aPosition 3D position of the voxel within the scene.
 * \param aViewPosition 3D position of the viewer or camera.
 * \param aColor Color of the volume at the voxel, representing its absorption characteristics.
 * \param aLightSources Pointer to an array of light sources influencing the voxel.
 * \param aFactorReflection Reflection factor applied to the sampled reflection texture value.
 * \param surfaceNormal Surface normal at the voxel position, precomputed for efficient shading.
 * \param aShininess Shininess exponent for the Blinn-Phong model, controlling the highlight size.
 * \param aScatteringWeight Weight between Blinn-Phong reflection and HG scattering components,
 *                          where 0 indicates full reflection and 1 indicates full scattering.
 * \param aHgAsymmetry Asymmetry factor \( g \) in the HG phase function, controlling forward vs.
 *                     backward scattering characteristics.
 * 
 * \return Computed color at the voxel based on illumination, reflection, and scattering.
 */
__device__ float3 shade(const float3 &aSamplePosition, const float3 &aPosition, 
                        const float3 &aViewPosition, const float3 &aColor, 
                        vr::LightSource *aLightSources, const float aFactorReflection, 
                        const float3 &surfaceNormal, const float aShininess, 
                        const float aScatteringWeight, const float aHgAsymmetry) {
  const float factorReflection = aFactorReflection;
  float3 result = make_float3(0.0f);

  for (size_t i = 0; i < c_numLightSources; ++i) {
    vr::LightSource lightSource = aLightSources[i];

    // Calculate light direction and distance
    float3 lightDir = lightSource.position - aPosition;
    float lightDistanceSquared = dot(lightDir, lightDir);
    lightDir = normalize(lightDir);

    // Apply attenuation unless intensity is -1 (indicating diffuse lighting)
    float attenuation = (lightSource.intensity == -1.0f) ? 1.0f : lightSource.intensity / (lightDistanceSquared + 1e-6f);

    // Calculate view direction and half-vector for Blinn-Phong
    float3 viewDir = normalize(aViewPosition - aPosition);
    float3 halfVector = normalize(lightDir + viewDir);

    // Reflection
    float3 reflectionComponent = make_float3(0.0f, 0.0f, 0.0f);
    if (aScatteringWeight < 1.0f) {
      // Blinn-Phong Diffuse and Specular Components
      float diffuseFactor = max(dot(surfaceNormal, lightDir), 0.0f);
      
      // Clamp specular factor to avoid sharp artifacts
      float specularFactor = pow(max(dot(surfaceNormal, halfVector), 0.0f), aShininess);
      // specularFactor = min(specularFactor, 1.0f);  // Clamp to [0, 1] for stability

      float3 diffuseComponent = diffuseFactor * lightSource.color * aColor;
      float3 specularComponent = specularFactor * lightSource.color;

      // Reflection texture lookup
      float reflection = factorReflection * tex3D(tex_reflection, aSamplePosition.x, aSamplePosition.y, aSamplePosition.z);

      // Combine Blinn-Phong reflection with reflection texture
      reflectionComponent = (1.0f - aScatteringWeight) * (diffuseComponent + specularComponent) * reflection;
    }

    // Scattering
    float phase = (phase_functions[dc_activePhaseMethod])(lightDir, viewDir, surfaceNormal, aHgAsymmetry);

    // Scattering component based on phase function
    float3 scatteringComponent = aScatteringWeight * phase * lightSource.color * aColor;

    // Accumulate the result for all light sources with attenuation applied at the end
    result += (scatteringComponent + reflectionComponent) * attenuation;
  }

  return result;
}


/*! \fn void d_render(float *d_aOutput,  const vr::RenderOptions aOptions,
         const float3 aColor, const vr::LightSource * aLightSources,
         const float3 aGradientStep)
 *  \brief performs raycasting on the device
 *  \param d_aOutput device pointer of the computed 2D output
 *  \param aOptions options of the rendering process
 * 	\param aColor the color the rendered volume absorbs
 * 	\param aLightSources pointer to all light sources
 * 	\param aGradientStep step size to a neighbor voxel
 */
__global__ void d_render(float *d_aOutput, const vr::RenderOptions aOptions,
                         const float3 aColor,
                         const vr::LightSource *aLightSources,
                         const float3 aGradientStep) {
  const uint x = blockIdx.x * blockDim.x + threadIdx.x;
  const uint y = blockIdx.y * blockDim.y + threadIdx.y;

  // return if out of image bound
  if ((x >= aOptions.image_width) || (y >= aOptions.image_height))
    return;

  vr::LightSource *lightSources =
      const_cast<vr::LightSource *>(aLightSources);
  const float tstep = aOptions.tstep;
  const float opacityThreshold = aOptions.opacity_threshold;
  const float factorAbsorption = aOptions.factor_absorption;
  const float factorEmission = aOptions.factor_emission;
  const float factorReflection = aOptions.factor_reflection;

  const float3 boxMin = aOptions.boxmin;
  const float3 boxMax = aOptions.boxmax;

  // 2D image plane in [-1,1]
  float u = (x / (float)aOptions.image_width) * 2.0f - 1.0f;
  float ratio = aOptions.image_height / (float)aOptions.image_width;
  float v = (y / (float)aOptions.image_height) * 2.0f * ratio - 1.0f * ratio;

  // calculate eye ray in world space
  vr::Ray eyeRay;

  // box factor
  const float3 boxScale = 1.f / (boxMax - boxMin);

  // translate into factor
  const float cameraXOffset = aOptions.rotation_matrix.m[3].x;
  const float focalLength = aOptions.rotation_matrix.m[3].y;
  const float objectDistance = aOptions.rotation_matrix.m[3].z;

  // in case of 3D rendering we have an x offset [Off-axis]
  const float3 xVector = aOptions.rotation_matrix.m[0];
  const float3 yVector = aOptions.rotation_matrix.m[1];
  const float3 zVector = aOptions.rotation_matrix.m[2];
  const float3 vCameraOffset = (cameraXOffset * xVector);

  // Ray properties
  eyeRay.origin = vCameraOffset + (-1 * objectDistance * zVector);

  eyeRay.direction =
      normalize(u * normalize(xVector) + v * yVector + focalLength * zVector);

  // find intersection with box
  float tnear(0), tfar(0);
  int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

  // if (hit)
  //    printf("near: %f, far: %f\n", tnear, tfar);

  if (!hit)
    return;
  if (tnear < 0.0f)
    tnear = 0.0f; // clamp to near plane

  // march along ray from front to back, accumulating color
  float4 sum = make_float4(0.0f);
  float t = tnear;
  float3 pos = eyeRay.origin + eyeRay.direction * tnear;
  float3 step = eyeRay.direction * tstep;

  // map step to [0, 1] coordinates
  float3 pos_sample_old = make_float3(0.f);
  while (true) {
    // map position to [0, 1] coordinates
    float3 pos_sample = (pos - boxMin) * boxScale;

    // ################
    // ### sampling ###
    // ################

    // read from 3D texture and apply several factor factor
    float emission = factorEmission * tex3D(getTexture(d_idxEmmission), pos_sample.x,
                                           pos_sample.y, pos_sample.z);
    float absorption = factorAbsorption * tex3D(getTexture(d_idxAbsorption), pos_sample.x,
                                               pos_sample.y, pos_sample.z);

    float3 sample = make_float3(emission);

    // ###############################
    // ### illumination & Coloring ###
    // ###############################

    float dx = tstep;
    float alpha = 1.0f - __expf(-absorption * dx);

    // apply color
    float ds = tstep;
    float3 colored = sample * ds * aColor;

    // Calculate surface normal based on the gradient
    const float3 surfaceNormal =
      -1.0f * normalize((gradient_functions[dc_activeGradientMethod])(
               pos_sample, pos, aGradientStep, boxMin, boxMax, boxScale));

    // compute pixel value
    float3 illumination =
        shade(pos_sample, pos, eyeRay.origin, aColor,
              lightSources, factorReflection, surfaceNormal,
              aOptions.shininess, aOptions.scattering_weight, aOptions.hg_asymmetry);

    float3 illuminated = colored + illumination;

    float4 shaded =
        make_float4(illuminated.x, illuminated.y, illuminated.z, alpha);

    // ###################
    // ### compositing ###
    // ###################

    // alpha-blending
    // pre-multiply alpha
    shaded.x *= shaded.w;
    shaded.y *= shaded.w;
    shaded.z *= shaded.w;

    // "under" operator for front-to-back blending
    sum = (1.0f - sum.w) * (shaded) + sum;

    // exit early if opaque
    if (sum.w > opacityThreshold)
      break;

    t += tstep;
    if (t > tfar)
      break;

    pos += step;
  }

  // write in image structure
  uint size = aOptions.image_width * aOptions.image_height;

  // linear matlab conform memory layout (column-major)
  // descibed on:
  // https://eli.thegreenplace.net/2015/memory-layout-of-multi-dimensional-arrays/
  uint k = x * aOptions.image_height + y;

  // write output in RBG
  d_aOutput[k] = sum.x;
  d_aOutput[k + size] = sum.y;
  d_aOutput[k + size * 2] = sum.z;
}


namespace vr {

/*! \fn hipArray * createTextureFromVolume(
                    texture<VolumeDataType, hipTextureType3D, hipReadModeElementType> &aTex,
                    const Volume &aVolume, hipArray *d_aArray, const bool aAllocateMemory) 
 *  \brief creates a texture based on a Volume
 *  \param aTex texture that is used to perform lookups
 *  \param aVolume volume that is copied to the device
 * 	\param d_aArray device array where the data are stored in/copied to
 *  \param aAllocateMemory if set to true, memory will be allocated and data copied from host to device
 * 	\return device pointer of the device array
 */
hipArray * createTextureFromVolume(
    texture<VolumeDataType, hipTextureType3D, hipReadModeElementType> &aTex,
    const Volume &aVolume, hipArray *d_aArray, const bool aAllocateMemory) {
  // if volume was refreshed or first render
  // bool allocateMemory = (aVolume.last_update > aTimeLastMemSync) || (aTimeLastMemSync == 0);

  // only allocate memory and copy data to GPU if required
  if (d_aArray == 0 || aAllocateMemory) {
    HANDLE_ERROR(hipMalloc3DArray(&d_aArray, &channelDesc, aVolume.extent));

    // copy data to 3D d_array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr(
      aVolume.data, aVolume.extent.width * sizeof(VolumeDataType),
      aVolume.extent.width, aVolume.extent.height);
    copyParams.dstArray = d_aArray;
    copyParams.extent = aVolume.extent;
    copyParams.kind = hipMemcpyHostToDevice;
    HANDLE_ERROR(hipMemcpy3D(&copyParams));
  }

  // set texture parameters
  aTex.normalized = true; // access with normalized texture coordinates
  aTex.filterMode = hipFilterModeLinear;     // linear interpolation
  aTex.addressMode[0] = hipAddressModeClamp; // clamp texture coordinates
  aTex.addressMode[1] = hipAddressModeClamp;
  aTex.addressMode[2] = hipAddressModeClamp;

  // bind d_aArray to 3D texture
  HANDLE_ERROR(hipBindTextureToArray(aTex, d_aArray, channelDesc));

  return d_aArray;
}

/*! \fn void freeCudaGradientBuffers(
              hipArray * d_aGradientXArray,
              hipArray * d_aGradientYArray,
              hipArray * d_aGradientZArray
            )
 *  \param d_aGradientXArray device memory address to the x-gradient
 *  \param d_aGradientYArray device memory address to the y-gradient
 *  \param d_aGradientZArray device memory address to the z-gradient
 *  \brief removes the gradient volumes from the device memory
 */
void freeCudaGradientBuffers(
  hipArray * d_aGradientXArray,
  hipArray * d_aGradientYArray,
  hipArray * d_aGradientZArray
) {
  HANDLE_ERROR(hipFreeArray(d_aGradientXArray));
  HANDLE_ERROR(hipFreeArray(d_aGradientYArray));
  HANDLE_ERROR(hipFreeArray(d_aGradientZArray));
}

/*! \fn void render_kernel(float* d_aOutput, const dim3& block_size,
                           const dim3& grid_size, const RenderOptions& aOptions,
                           const float3& aColor, const float3& aGradientStep)
 *  \brief starts the ray casting on the device
 * 	\param d_aOutput device pointer of the computed 2D output
 *  \param block_size CUDA block size
 * 	\param grid_size CUDA grid size
 *  \param aOptions options of the rendering process
 * 	\param aColor the color the rendered volume absorbs
 * 	\param aGradientStep step size to a neighbor voxel
 */
void render_kernel(float *d_aOutput, const dim3 &block_size,
                   const dim3 &grid_size, const RenderOptions &aOptions,
                   const float3 &aColor, const float3 &aGradientStep) {
  d_render<<<grid_size, block_size>>>(d_aOutput, aOptions, aColor,
                                      d_lightSources, aGradientStep);
}

/*! \fn void copyLightSources(const LightSource *aLightSources,
                      const size_t aNumOfLightSources)
 *  \brief copy light sources to device
 * 	\param aLightSources pointer to all light sources
 * 	\param aNumOfLightSources number of light sources
 */
void copyLightSources(const LightSource *aLightSources,
                      const size_t aNumOfLightSources) {
  size_t size(aNumOfLightSources * sizeof(LightSource));
  HANDLE_ERROR(hipMalloc((void **)&d_lightSources, size));
  HANDLE_ERROR(
      hipMemcpy(d_lightSources, aLightSources, size, hipMemcpyHostToDevice));

  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(c_numLightSources), &aNumOfLightSources,
                                  sizeof(size_t)));
}

/*! \fn void setGradientMethod(const vr::GradientMethod)
 *  \brief set gradientMethod
 * 	\param aMethod kind of gradient method used while rendering
 */
void setGradientMethod(const vr::GradientMethod aMethod) {
  HANDLE_ERROR(
    hipMemcpyToSymbol(HIP_SYMBOL(dc_activeGradientMethod), &aMethod, sizeof(enum vr::GradientMethod))
  );
}

/*! \fn void setPhaseMethod(const vr::PhaseMethod)
 *  \brief set scatter method for rendering
 *  \param aMethod specifies the scattering method to use (e.g., compute or lookup)
 */
void setPhaseMethod(const vr::PhaseMethod aMethod) {
    HANDLE_ERROR(
        hipMemcpyToSymbol(HIP_SYMBOL(dc_activePhaseMethod), &aMethod, sizeof(enum vr::PhaseMethod))
    );
}

/*! \fn hipArray * referenceTexture(
            texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType>& aTexture, 
            hipArray* d_aArray, const vr::VolumeType& d_aIdx, const vr::VolumeType aTypeAssigned)
 *  \brief reference a texture, using the mechanism undelying getTexture
 *  \param aTexture texture which should be re-referenced
 * 	\param d_aArray array pointing to the device memory undelying the texture
 *  \param d_aIdx volume type id the texture has
 * 	\param aTypeAssigned the volume type id the texture should refer to
 *  \return device address of texture
 */
hipArray * referenceTexture(
    texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType>& aTexture, 
    hipArray* d_aArray, const vr::VolumeType& d_aIdx, const vr::VolumeType aTypeAssigned) {
  
  // clear memory
  if (d_aArray != 0) {
    HANDLE_ERROR(hipUnbindTexture(aTexture));
    d_aArray = 0;
  }
  
  // just reference to the same device variable
  // HANDLE_ERROR(hipBindTextureToArray(aTexture, d_aArrayRef, channelDesc));
  HANDLE_ERROR(
    hipMemcpyToSymbol(HIP_SYMBOL(d_aIdx), &aTypeAssigned, sizeof(enum vr::VolumeType))
  );

  return d_aArray;
}

/*! \fn hipArray * syncVolume(
          texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType>& aTexture,
          hipArray* &d_aArray, const Volume& aVolume, const bool aAllocateMemory)
 *  \brief sync volume with device (if required) and setup texture
 * 	\param aTexture texture which should be synced
 * 	\param d_aArray array pointing to the device memory undelying the texture
 * 	\param aVolume volume data which should be synched onto the device
 *  \param aAllocateMemory if set to true, memory will be allocated and data copied from host to device
 */
hipArray * syncVolume(
    texture<vr::VolumeDataType, hipTextureType3D, hipReadModeElementType>& aTexture,
    hipArray* d_aArray, const Volume& aVolume, const bool aAllocateMemory) {
  // clear memory
  if (d_aArray != 0 || aAllocateMemory) {
    HANDLE_ERROR(hipUnbindTexture(aTexture));
    HANDLE_ERROR(hipFreeArray(d_aArray));
    d_aArray = 0;
  }

  d_aArray = createTextureFromVolume(aTexture, aVolume, d_aArray, aAllocateMemory);

  return d_aArray;
}

/*! \fn hipArray* setPhaseTexture(const Volume &aVolume, 
                                          hipArray * d_aPhase, 
                                          const uint64_t aTimeLastMemSync)
 *  \brief copies phase volume from host to device
 *  \param aVolume phase volume
 *  \param d_aPhase array pointing to the device memory
 *  \param aTimeLastMemSync timestamp on which the last rendering took place
 */
hipArray * setPhaseTexture(const Volume &aVolume, 
                                   hipArray * d_aPhase, 
                                   const uint64_t aTimeLastMemSync) {

  bool allocateMemory = (aVolume.last_update > aTimeLastMemSync) || (aTimeLastMemSync == 0);
  
  return syncVolume(tex_phase, d_aPhase, aVolume, allocateMemory);
}

/*! \fn setGradientTextures(const Volume &aDx, 
                         const Volume &aDy,
                         const Volume &aDz, 
                         hipArray * &ptr_d_volumeDx,
                         hipArray * &ptr_d_volumeDy,
                         hipArray * &ptr_d_volumeDz,
                         const uint64_t aTimeLastMemSync)
 *  \brief copies gradient volumes from host to device 
 *  \param aDx volume of gradient in x direction 
 *  \param aDy volume of gradient in y direction 
 *  \param aDz volume of gradient in z direction
 */
void setGradientTextures(const Volume &aDx, 
                         const Volume &aDy,
                         const Volume &aDz, 
                         hipArray * &ptr_d_volumeDx,
                         hipArray * &ptr_d_volumeDy,
                         hipArray * &ptr_d_volumeDz,
                         const uint64_t aTimeLastMemSync) {
  bool allocateMemoryDx = (aDx.last_update > aTimeLastMemSync) || (aTimeLastMemSync == 0);
  bool allocateMemoryDy = (aDy.last_update > aTimeLastMemSync) || (aTimeLastMemSync == 0);
  bool allocateMemoryDz = (aDz.last_update > aTimeLastMemSync) || (aTimeLastMemSync == 0);

  ptr_d_volumeDx = syncVolume(tex_gradientX, ptr_d_volumeDx, aDx, allocateMemoryDx);
  ptr_d_volumeDy = syncVolume(tex_gradientY, ptr_d_volumeDy, aDy, allocateMemoryDy);
  ptr_d_volumeDz = syncVolume(tex_gradientZ, ptr_d_volumeDz, aDz, allocateMemoryDz);

  // assign gradient_function
  vr::GradientMethod tmp = gradientLookup;
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dc_activeGradientMethod), &tmp,
                                  sizeof(enum vr::GradientMethod)));
}

/*! \fn void syncWithDevice(const Volume &aVolumeEmission, const Volume &aVolumeAbsorption,
                    const Volume &aVolumeReflection, const uint64_t aTimeLastMemSync,
                    hipArray * &d_aVolumeEmission, hipArray * &d_aVolumeAbsorption, 
                    hipArray * &d_aVolumeReflection)
 * \brief Copies volume data to device and binds textures to the appropriate data.
 *         Data of one volume can be assigned to multiple textures and won't be copied to device, 
 *         if nothing had been changed since the last rendering.
 * \param aVolumeEmission volume for emission
 * \param aVolumeAbsorption volume for absorption
 * \param aVolumeReflection volume for reflection
 * \param aTimeLastMemSync timestamp on which the last rendering took place
 * \param d_aVolumeEmission array pointing to the device memory of the emission volume
 * \param d_aVolumeAbsorption array pointing to the device memory of the absorption volume
 * \param d_aVolumeReflection array pointing to the device memory of the reflection volume
 */
void syncWithDevice(const Volume &aVolumeEmission, const Volume &aVolumeAbsorption,
                    const Volume &aVolumeReflection, const uint64_t aTimeLastMemSync,
                    hipArray * &d_aVolumeEmission, hipArray * &d_aVolumeAbsorption, 
                    hipArray * &d_aVolumeReflection) {
  // similarities of volumes
  const bool simEmAb = (aVolumeEmission == aVolumeAbsorption);
  const bool simEmRe = (aVolumeEmission == aVolumeReflection);
  const bool simAbRe = (aVolumeAbsorption == aVolumeReflection);

  // update required
  const bool reqUpdateEm = (aVolumeEmission.last_update > aTimeLastMemSync) || (aTimeLastMemSync == 0);
  const bool reqUpdateAb = (aVolumeAbsorption.last_update > aTimeLastMemSync) || (aTimeLastMemSync == 0);
  const bool reqUpdateRe = (aVolumeReflection.last_update > aTimeLastMemSync) || (aTimeLastMemSync == 0);

  // save status
  bool updatedEm = false;
  bool updatedAb = false;
  bool updatedRe = false;

#ifdef DEBUG
  mexPrintf("Emission %d\n", reqUpdateEm);
  mexPrintf("Absorption %d\n", reqUpdateAb);
  mexPrintf("Reflection %d\n", reqUpdateRe);
#endif

  // conditionally update GPU memory and textures in order to save bandwidth
  if (reqUpdateEm) {
    if (!updatedEm) {
      d_aVolumeEmission = syncVolume(tex_emission, d_aVolumeEmission, aVolumeEmission, updatedEm);
      updatedEm = true;
    }

    if (simEmRe && !updatedRe) {
      d_aVolumeReflection = referenceTexture(tex_reflection, d_aVolumeReflection,
                   d_idxReflection, vr::VolumeType::emission);
      updatedRe = true;

#ifdef DEBUG
  mexPrintf("Emission = Reflection\n");
  mexPrintf("setup Reflection\n");
#endif
    }

    if (simEmAb && !updatedAb) {
      d_aVolumeAbsorption = referenceTexture(tex_absorption, d_aVolumeAbsorption,
                   d_idxAbsorption, vr::VolumeType::emission);
      updatedAb = true;

#ifdef DEBUG
  mexPrintf("Emission = Absorption\n");
  mexPrintf("setup Reflection: %d\n", updatedAb);
#endif
    }
  }

  if (reqUpdateAb) {
    if (!updatedAb) {
      d_aVolumeAbsorption =
          syncVolume(tex_absorption, d_aVolumeAbsorption, aVolumeAbsorption, updatedAb);
      updatedAb = true;

#ifdef DEBUG
  mexPrintf("Synced Volume Absorption\n");
#endif
    }

    if (simAbRe && !updatedRe) {
      d_aVolumeReflection = referenceTexture(tex_reflection, d_aVolumeReflection,
                   d_idxReflection, vr::VolumeType::absorption);
      updatedRe = true;

#ifdef DEBUG
  mexPrintf("Absorption = Reflection\n");
  mexPrintf("setup Reflection: %d\n", updatedRe);
#endif

    }

    if (simEmAb && !updatedEm) {
      d_aVolumeAbsorption = referenceTexture(tex_emission, d_aVolumeAbsorption,
                   d_idxEmmission, vr::VolumeType::absorption);
      updatedEm = true;

#ifdef DEBUG
  mexPrintf("Absorption = Emission\n");
  mexPrintf("setup Emission: %d\n", updatedEm);
#endif

    }
  }

  if (reqUpdateRe) {
    if (!updatedRe) {
      d_aVolumeReflection = 
          syncVolume(tex_reflection, d_aVolumeReflection, aVolumeReflection, reqUpdateRe);
      updatedRe = true;

#ifdef DEBUG
  mexPrintf("Synced Volume Reflection\n");
#endif
    }

    if (simAbRe && !updatedAb) {
      d_aVolumeAbsorption = referenceTexture(tex_absorption, d_aVolumeAbsorption,
                   d_idxAbsorption, vr::VolumeType::reflection);
      updatedAb = true;

#ifdef DEBUG
  mexPrintf("Reflection = Absorption\n");
  mexPrintf("setup Absorption: %d\n", updatedAb);
#endif

    }

    if (simEmAb && !updatedEm) {
      d_aVolumeEmission = referenceTexture(tex_emission, d_aVolumeEmission,
                   d_idxEmmission, vr::VolumeType::reflection);
      updatedEm = true;

#ifdef DEBUG
  mexPrintf("Reflection = Emission\n");
  mexPrintf("setup Emission: %d\n", updatedEm);
#endif
    }
  }

  // no further check is necessary
  return;
};
} // namespace vr
#endif // #ifndef _VOLUMERENDER_KERNEL_CU_
